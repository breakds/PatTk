#include "hip/hip_runtime.h"
/*********************************************************************************
 * File: BP_CUDA.cu
 * Description: Belief Propagation for Ising Model, CUDA parallelized version
 * by BreakDS, @ University of Wisconsin-Madison, Fri Aug 31 15:25:19 CDT 2012
 *********************************************************************************/

#include "BP_CUDA.h"
#include <hip/hip_runtime.h>
#include "../utils/cuda_aux.hpp"



// Temporary Constants
#define PATCH_SIDE 17

__device__ __constant__ int g_inc[4];
  

namespace optimize_cuda
{

  /*
  __host__ __device__ inline float dist( const float *a, const float *b,
                                int __attribute__((__unused__)) dim,
                                int direction = -1 )
  {
    const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
    float tmp;
    tmp = ( a[0] > b[0] ) ? ( a[0] - b[0] ) : ( b[0] - a[0] );


    // [0] = image index
    tmp = ( a[0] > b[0] ) ? ( a[0] - b[0] ) : ( b[0] - a[0] );
    if ( tmp > 1.0 ) return 150000.0;

    // [1], [2] = dy, dx (rotation representation)
    tmp = ( a[1] > b[1] ) ? ( a[1] - b[1] ) : ( b[1] - a[1] );
    tmp += ( a[2] > b[2] ) ? ( a[2] - b[2] ) : ( b[2] - a[2] );
    if ( tmp > 1.0 ) {
      return 150000.0;
    }

      
    // [4],[5] = spatial distance
    // Should be compensated by the rotation (dy,dx)
    // Note that dy = b[1] dx = b[2] by definition
    float ay(a[4]), ax(a[5]);
    if ( 0 == direction ) {
      ay += b[2];
      ax -= b[1];
    } else if ( 1 == direction ) {
      ay += b[1];
      ax -= b[2];
    } else if ( 2 == direction ) {
      ay += b[2];
      ax += b[1];
    } else if ( 3 == direction ) {
      ay -= b[1];
      ax += b[2];
    }
      
    tmp = ( ay > b[4] ) ? ( ay - b[4] ) : ( b[4] - ay );
    tmp += ( ax > b[5] ) ? ( ax - b[5] ) : ( b[5] - ax );
    if ( tmp > PATCH_SIDE ) {
      return 150000.0;
    }
      
    // [1],[2] = spatial distance
    
    float sum = tmp * coeff[4];
    
    for ( int i=1; i<4; i++ ) {
      sum += ( ( a[i] > b[i] ) ? ( a[i] - b[i] ) : ( b[i] - a[i] ) ) * coeff[i];
    }
    return sum;
  }
  */
  

  __host__ __device__ inline float dist( const float a0, const float a1,
                                         const float a2, const float a3,
                                         const float a4, const float a5,
                                         const float b0, const float b1,
                                         const float b2, const float b3,
                                         const float b4, const float b5,
                                         int __attribute__((__unused__)) dim,
                                         int direction = -1 )
  {
    const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
    float tmp;

    // [0] = image index
    tmp = fabsf( b0 - a0 );
    if ( tmp > 1.0 ) return 150000.0;

    // [1], [2] = dy, dx (rotation representation)
    tmp = fabsf( a1 - b1 ) + fabsf( a2 - b2  );
    if ( tmp > 1.0 ) {
      return 150000.0;
    }

      
    // [4],[5] = spatial distance
    // Should be compensated by the rotation (dy,dx)
    // Note that dy = b1 dx = b2 by definition

    float ay(a4), ax(a5);

    if ( 0 == direction ) {
      ay += b2;
      ax -= b1;
    } else if ( 1 == direction ) {
      ay += b1;
      ax -= b2;
    } else if ( 2 == direction ) {
      ay += b2;
      ax += b1;
    } else if ( 3 == direction ) {
      ay -= b1;
      ax += b2;
    }
    
    
    
      
    tmp = fabsf( ay - b4 ) + fabsf( ax - b5 );
    if ( tmp > PATCH_SIDE ) {
      return 150000.0;
    }
      
    // [1],[2] = spatial distance
    
    float sum = tmp * coeff[4];

    sum += fabsf( b1 - a1 ) * coeff[1];
    sum += fabsf( b2 - a2 ) * coeff[2];
    sum += fabsf( b3 - a3 ) * coeff[3];

    
    return sum;
  }
  


  __global__ void NormalizeMessages_float_agent( const int agentNum,
                                                 float *msg,
                                                 int K )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      float avg = 0.0f;
      for ( int k=0; k<K; k++ ) {
        avg += msg[ idx * K + k ];
        // avg += msg[ k * agentNum + idx ];
      }
      avg /= K;
      for ( int k=0; k<K; k++ ) {
        msg[ idx * K + k ] -= avg;
        // msg[ k * agentNum + idx ] -= avg;
      }
    }
  }

  
  
  __global__ void UpdateResult_float_agent( const int agentNum, const float *D,
                                            const float* msg,
                                            int *result, int K, int dim, float lambda )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if ( idx < agentNum ) {
      result[idx] = 0;
      float min = 0.0f;
      for ( int k=0; k<K; k++ ) {
        float sum = D[idx*K+k];
        for ( int dir=0; dir<4; dir++ ) {
          sum += msg[ ( dir * agentNum + idx ) * K + k ];
          // sum += msg[ ( k * 4  + dir ) * agentNum + idx ];
        }

        if ( 0 == k ) {
          min = sum;
        } else if ( sum < min ) {
          min = sum;
          result[idx] = k;
        }
      }
    }
  }

  

  double UpdateEnergy( const float *D, const float *label, int height, int width, int K,
                       int dim, float lambda, int *result )
  {

    // constants
    const int inc[4] = { -width, -1, width, 1 };
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};
    
    
    double energy = 0.0;
    int i = 0;
    const float *labelp = label;
    for ( int y=0; y<height; y++ ) {
      for ( int x=0; x<width; x++ ) {
        energy += D[i*K+result[i]];
        
        // UP:
        int d = 0;
        if ( y > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          // energy += dist( lp0, lp1, dim, d ) * lambda;
          energy += dist( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                          lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5],
                          dim, d ) * lambda;
        }
        

        // LEFT:
        d = 1;
        if ( x > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          // energy += dist( lp0, lp1, dim, d ) * lambda;
          energy += dist( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                          lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5],
                          dim, d ) * lambda;
        }

        i++;
        labelp += K * dim;
      }
    }

    return energy;
  }

  /*
  __global__ void LoopyBP_round1( int agentNum, // number of agents needes
                                  int iter,
                                  float *D,
                                  float* msg,
                                  float *distance,
                                  int* begins,
                                  int inc,
                                  float *h_buf,
                                  int K,
                                  int dir,
                                  int area )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      int k = threadIdx.y;
      int opp = (dir+2) & 3;
      int pos = k * agentNum + idx;
      int pixel = begins[idx] + inc * iter;
      float tmp = D[pixel * K + k];
      for ( int j=0; j<4; j++ ) {
        if ( j != opp ) {
          // h_buf[pos] += msg[ j * area * K + pixel * K + k];
          tmp += msg[ ( k * 4 + j ) * area + pixel ];
        }
      }
      h_buf[pos] = tmp;
    }
  }
  
  __global__ void LoopyBP_round2( int agentNum, // number of agents needes
                                  int iter,
                                  float *D,
                                  float* msg,
                                  float *distance,
                                  int* begins,
                                  int inc,
                                  float *h_buf,
                                  int K,
                                  float lambda,
                                  int dir,
                                  int area )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      int k = threadIdx.y;
      int pos = k * agentNum + idx;
      int pixel = begins[idx] + inc * iter;
      float min = distance[ ( k * 4  + dir ) * area + pixel ] * lambda + h_buf[pos];
      for ( int k0=1; k0<K; k0++ ) {
        float value = distance[ ( ( k0 * K + k ) * 4 + dir ) * area + pixel ] * lambda +
          h_buf[pos + k0 * agentNum];
        if ( value < min ) min = value;
      }
      // msg[ dir * area * K + ( pixel + inc ) * K + k ] = min;
      msg[ ( k * 4 + dir ) * area + pixel + inc ] = min;
    }
  }
  
  */

  
  // Kernel Function for loopy belief propagation (agentNum = width/height)
  __global__ void LoopyBP_agent_float( int agentNum, // number of agents needes
                                       float *D,
                                       float* msg,
                                       float* distance,
                                       int *begins,
                                       int *ends,
                                       int K,
                                       float lambda,
                                       int dim,
                                       int dir,
                                       int inc,
                                       int incK,
                                       int incDim,
                                       int area )
  {
    
    extern __shared__ float buf[];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      float *h = buf;
      // float *l0 = buf + blockDim.x * K + threadIdx.x * dim;
      // float *l1 = buf + blockDim.x * ( K + dim ) + threadIdx.x * dim;
      // agent initialization
      int opp = (dir+2) & 3;
      //      float *h = buf_h + idx * K;
      float *Dp = D + begins[idx] * K;
      
      for ( int i=begins[idx]; i!=ends[idx]; i+=inc, Dp+=incK ) {
        
        for ( int k=0; k<K; k++ ) {
          h[ k * blockDim.x + threadIdx.x ] = Dp[k];
          for ( int j=0; j<4; j++ ) {
            if ( j != opp ) {
              h[ k * blockDim.x + threadIdx.x ] += msg[ j * area * K + i * K + k ];
              // h[ k * blockDim.x + threadIdx.x ] += msg[ ( k * 4 + j ) * area + i ];
            }
          }
        }

        for ( int k=0; k<K; k++ ) {
          float min = distance[ ( k * 4 + dir ) * area  + i ] * lambda + h[ threadIdx.x ];
          
          for ( int k0=1; k0<K; k0++ ) {
            float value = distance[ ( ( k0 * K + k ) * 4 + dir ) * area  + i ] * lambda +
              h[ k0 * blockDim.x + threadIdx.x];
            if ( value < min ) min = value;
          }
          msg[ dir * area * K + ( i + inc ) * K + k ] = min;
          // msg[ ( k * 4 + dir ) * area + i + inc ] = min;
        }

      } // end for i
    }
  }

  __global__ void LoopyBP_agent_float_v2( int agentNum, // number of agents needes
                                          float *D,
                                          float* msg,
                                          float* distance,
                                          int *begins,
                                          int *ends,
                                          int K,
                                          float lambda,
                                          int dim,
                                          int dir,
                                          int inc,
                                          int incK,
                                          int incDim,
                                          int area )
  {
    
    extern __shared__ float buf[];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      int k = threadIdx.y;
      float *h = buf;
      // float *l0 = buf + blockDim.x * K + threadIdx.x * dim;
      // float *l1 = buf + blockDim.x * ( K + dim ) + threadIdx.x * dim;
      // agent initialization
      int opp = (dir+2) & 3;
      //      float *h = buf_h + idx * K;
      float *Dp = D + begins[idx] * K;
      
      for ( int i=begins[idx]; i!=ends[idx]; i+=inc, Dp+=incK ) {
        
        h[ k * blockDim.x + threadIdx.x ] = Dp[k];
        for ( int j=0; j<4; j++ ) {
          if ( j != opp ) {
            h[ k * blockDim.x + threadIdx.x ] += msg[ j * area * K + i * K + k ];
            // h[ k * blockDim.x + threadIdx.x ] += msg[ ( k * 4 + j ) * area + i ];
          }
        }
        __syncthreads();

        float min = distance[ ( k * 4 + dir ) * area  + i ] * lambda + h[ threadIdx.x ];


        for ( int k0=1; k0<K; k0++ ) {
          __syncthreads();
          float value = distance[ ( ( k0 * K + k ) * 4 + dir ) * area  + i ] * lambda +
            h[ k0 * blockDim.x + threadIdx.x];
          if ( value < min ) min = value;
        }
        msg[ dir * area * K + ( i + inc ) * K + k ] = min;
        // msg[ ( k * 4 + dir ) * area + i + inc ] = min;

      } // end for i
    }
  }


  __global__ void Precomputing_agent_float( int area,
                                            float *label,
                                            float *distance,
                                            int dim,
                                            int K )
  {
    int pixel = blockDim.x * blockIdx.x + threadIdx.x;
    if ( pixel < area ) {
      int dir = blockIdx.y;
      // int k0 = threadIdx.y;
      // int k1 = threadIdx.z;
      
      int idx = ( ( threadIdx.y * K + threadIdx.z ) * 4 + dir ) * area + pixel;
      // overflow checking
      if ( pixel + g_inc[dir] < 0 || pixel + g_inc[dir] >= area ) {
        return ;
      }
      
      float *lp0 = label + ( pixel * K + threadIdx.y ) * dim;
      float *lp1 = label + ( ( pixel + g_inc[dir] ) * K + threadIdx.z ) * dim;
      
      distance[idx] = dist( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                            lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5],
                            dim, dir );
    }
  }




  
  
  double LoopyBP( const float *D, // The distance term for each pixel, h x w
                  const float *label,
                  // the label vectors for each candidates of each pixel, h x w x K x dim
                  const int height,
                  const int width,
                  const int K,
                  const int dim,
                  int *result,
                  Options options, float* msgBuf )
  {
    const int inc[4] = { -width, -1, width, 1 };
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "g_inc"), inc, sizeof(int) * 4, 0, hipMemcpyHostToDevice ) );
    const int incK[4] = {-width*K,-K,width*K,K};
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};
    const int order[4] = {2,0,3,1}; // DOWN, UP, RIGHT, LEFT
    const int area = height * width;

    
    // Make sure that the message storage is allocated
    float *buf = msgBuf;
    if ( NULL == buf ) {
      // Message buffer is not provided externally
      buf = new float[4*width*height*K];
    }

    



    // Initialization of device memory
    // Data term array
    float *devD = NULL;
    hipMalloc( (void**) &devD, sizeof(float) * area * K );
    hipMemcpy( devD, D, sizeof(float) * area * K, hipMemcpyHostToDevice );

    // Labels array
    float *devLabel = NULL;
    hipMalloc( (void**) &devLabel, sizeof(float) * area * K * dim );
    hipMemcpy( devLabel, label, sizeof(float) * area * K * dim, hipMemcpyHostToDevice );
    
    // Messages
    // After these lines, msg[0] = up, msg[1] = left, msg[2] = right, msg[3] = down
    float *devMsg = NULL;
    hipMalloc( (void**) &devMsg, sizeof(float) * area * K * 4 );
    hipMemset( devMsg, 0, sizeof(float) * area * K * 4 );
    

    // begins and ends
    int longer = height > width ? height : width;
    int *devBegins = NULL;
    int *devEnds = NULL;
    int begins[longer];
    int ends[longer];
    hipMalloc( (void**) &devBegins, sizeof(int) * longer );
    hipMalloc( (void**) &devEnds, sizeof(int) * longer );

    // Result
    int *devResult = NULL;
    hipMalloc( (void**) &devResult, sizeof(int) * area * K );

    // h buffer
    // float *devH = NULL;
    // hipMalloc( (void**) &devH, sizeof(float) * longer * K );


    // Intermediate Distance Calculation
    float *devDistance = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devDistance, sizeof(float) * K * K * 4 * area ) );

    dim3 precompute_grid( ( area + 1 ) / 8, 4 );
    dim3 precompute_block( 8, K, K );
    


    // uint kernelTime;
    // cutCreateTimer(&kernelTime);
    // cutResetTimer(kernelTime);
    Precomputing_agent_float<<<precompute_grid, precompute_block>>>( area,
                                                                     devLabel,
                                                                     devDistance,
                                                                     dim,
                                                                     K );
    HANDLE_ERROR( hipDeviceSynchronize() );
    



    double energy(0);

    for ( int iter=0; iter<options.maxIter; iter++ ) {
      for ( int dirID=0; dirID<4; dirID++ ) {
        int dir = order[dirID];

        // fill in begins and ends
        int agentNum = ( 0 == ( dir & 1 ) ) ? width : height;
        int length = 0;
        if ( 0 == dir ) {
          // UP:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan + width * ( height - 1 );
            ends[scan] = scan;
            length = height - 1;
          }
        } else if ( 1 == dir ) {
          // LEFT:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = width * scan + width - 1;
            ends[scan] = width * scan;
            length = width - 1;
          }
        } else if ( 2 == dir ) {
          // DOWN:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan;
            ends[scan] = scan + width * ( height - 1 );
            length = height - 1;
          }
        } else if ( 3 == dir ) {
          // RIGHT:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan * width;
            ends[scan] = scan * width + width - 1;
            length = width - 1;
          }
        }
        
        hipMemcpy( devBegins, begins, sizeof(int) * agentNum, hipMemcpyHostToDevice );
        hipMemcpy( devEnds, ends, sizeof(int) * agentNum, hipMemcpyHostToDevice );
        


        // Alternative Calling
        // int blockSize = 16;
        // dim3 blockDim( blockSize, K );
        // for ( int i=0; i<length; i++ ) {
        //   LoopyBP_round1<<<(agentNum/blockSize)+1, blockDim>>>( agentNum,
        //                                                         i,
        //                                                         devD,
        //                                                         devMsg,
        //                                                         devDistance,
        //                                                         devBegins,
        //                                                         inc[dir],
        //                                                         devH,
        //                                                         K,
        //                                                         dir,
        //                                                         area );
        //   LoopyBP_round2<<<(agentNum/blockSize)+1, blockDim>>>( agentNum,
        //                                                         i,
        //                                                         devD,
        //                                                         devMsg,
        //                                                         devDistance,
        //                                                         devBegins,
        //                                                         inc[dir],
        //                                                         devH,
        //                                                         K,
        //                                                         options.lambda,
        //                                                         dir,
        //                                                         area );
        // }

        // call Kernel Function 1
        int blockSize = 8;
        dim3 blockDim( blockSize, K );
        int shMemSizePerBlock= blockSize * K * sizeof(float);
        LoopyBP_agent_float_v2<<<(agentNum/blockSize)+1,blockDim, shMemSizePerBlock>>>( agentNum, 
                                                                                        devD,
                                                                                        devMsg,
                                                                                        devDistance,
                                                                                        devBegins, 
                                                                                        devEnds,
                                                                                        K, 
                                                                                        options.lambda, 
                                                                                        dim, dir, inc[dir],
                                                                                        incK[dir], incDim[dir],
                                                                                        area );
        
        HANDLE_ERROR( hipDeviceSynchronize() );


        
        // hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost );

        // energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
        
        NormalizeMessages_float_agent<<<(width*height*4)/128+1,128>>>( width * height * 4,
                                                                       devMsg,
                                                                       K );
        if ( 1 <= options.verbose ) {

          UpdateResult_float_agent<<<(width*height+1)/64,64>>>( width * height,
                                                                devD,
                                                                devMsg,
                                                                devResult,
                                                                K,
                                                                dim,
                                                                options.lambda );
          
          hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost );
          energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
          printf( "Iteration %d: energy = %.5lf\n", iter, energy );
        }
        
        

      } // end for dir
      
    }

    UpdateResult_float_agent<<<(width*height+1)/64,64>>>( width * height,
                                                          devD,
                                                          devMsg,
                                                          devResult,
                                                          K,
                                                          dim,
                                                          options.lambda );

    hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost );
    energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
    printf( "Final energy = %.5lf\n", energy );

    // Free Cuda Memory
    if ( devD ) hipFree( devD );
    if ( devLabel ) hipFree( devLabel );
    if ( devMsg ) hipFree( devMsg );
    if ( devBegins ) hipFree( devBegins );
    if ( devEnds ) hipFree( devEnds );
    if ( devResult ) hipFree( devResult );
    if ( devDistance ) hipFree( devDistance );
    HANDLE_ERROR( hipDeviceSynchronize() );
    
    return energy;
  }
};



