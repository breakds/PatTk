#include "hip/hip_runtime.h"
/*********************************************************************************
 * File: BP_CUDA.cu
 * Description: Belief Propagation for Ising Model, CUDA parallelized version
 * by BreakDS, @ University of Wisconsin-Madison, Fri Aug 31 15:25:19 CDT 2012
 *********************************************************************************/


#include "BP_CUDA.h"
#include <hip/hip_runtime.h>
#include "../utils/cuda_aux.hpp"



// Temporary Constants
#define PATCH_SIDE 17

namespace optimize_cuda
{

  __host__ __device__ inline float dist( const float *a, const float *b,
                                int __attribute__((__unused__)) dim,
                                int direction = -1 )
  {
    const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
    float tmp;
    tmp = ( a[0] > b[0] ) ? ( a[0] - b[0] ) : ( b[0] - a[0] );


    // [0] = image index
    tmp = ( a[0] > b[0] ) ? ( a[0] - b[0] ) : ( b[0] - a[0] );
    if ( tmp > 1.0 ) return 150000.0;

    // [1], [2] = dy, dx (rotation representation)
    tmp = ( a[1] > b[1] ) ? ( a[1] - b[1] ) : ( b[1] - a[1] );
    tmp += ( a[2] > b[2] ) ? ( a[2] - b[2] ) : ( b[2] - a[2] );
    if ( tmp > 1.0 ) {
      return 150000.0;
    }

      
    // [4],[5] = spatial distance
    // Should be compensated by the rotation (dy,dx)
    // Not that dy = b[1] dx = b[2]
    float ay(a[4]), ax(a[5]);
    if ( 0 == direction ) {
      ay += b[2];
      ax -= b[1];
    } else if ( 1 == direction ) {
      ay += b[1];
      ax -= b[2];
    } else if ( 2 == direction ) {
      ay += b[2];
      ax += b[1];
    } else if ( 3 == direction ) {
      ay -= b[1];
      ax += b[2];
    }
      
    tmp = ( ay > b[4] ) ? ( ay - b[4] ) : ( b[4] - ay );
    tmp += ( ax > b[5] ) ? ( ax - b[5] ) : ( b[5] - ax );
    if ( tmp > PATCH_SIDE ) {
      return 150000.0;
    }
      
    // [1],[2] = spatial distance

    float sum = tmp * coeff[4];
    for ( int i=1; i<4; i++ ) {
      sum += ( ( a[i] > b[i] ) ? ( a[i] - b[i] ) : ( b[i] - a[i] ) ) * coeff[i];
    }
    return sum;
  }



  __global__ void NormalizeMessages_float_agent( const int agentNum,
                                                 float *msg,
                                                 int K )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      float avg = 0.0f;
      for ( int dir=0; dir<4; dir++ ) {
        for ( int k=0; k<K; k++ ) {
          avg += msg[ ( dir * agentNum + idx ) * K + k ];
        }
        avg /= K;
        for ( int k=0; k<K; k++ ) {
          msg[ ( dir * agentNum + idx ) * K + k ] -= avg;
        }
      }
    }
  }
  
  __global__ void UpdateResult_float_agent( const int agentNum, const float *D, const float *label,
                                            const float* msg,
                                            int *result, int K, int dim, float lambda )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if ( idx < agentNum ) {
      result[idx] = 0;
      float min = 0.0f;
      for ( int k=0; k<K; k++ ) {
        float sum = D[idx*K+k];
        for ( int dir=0; dir<4; dir++ ) {
          sum += msg[ ( dir * agentNum + idx ) * K + k ];
        }

        if ( 0 == k ) {
          min = sum;
        } else if ( sum < min ) {
          min = sum;
          result[idx] = k;
        }
      }
    }
  }

  

  /*
  double UpdateResult( const float* D, const float* label, float** msg, int* result, 
                       int K, int dim, int height, int width, float lambda )
                       {
  
    // constants
    const int inc[4] = { -width, -1, width, 1 };
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};


    // Update result
    const int area = height * width;
    const float *Dp = D;
    float *msgp[4];
    for ( int dir=0; dir<4; dir++ ) msgp[dir] = msg[dir];

    for ( int i=0; i<area; i++ ) {
      result[i] = 0;
      float min = 0;
      for ( int k=0; k<K; k++ ) {
        float sum = Dp[k];
        for ( int dir=0; dir<4; dir++ ) {
          sum += msgp[dir][k];
        }

        if ( 0 == k ) {
          min = sum;
        } else if ( sum < min ) {
          min = sum;
          result[i] = k;
        }
      }
      Dp += K;
      for ( int dir=0; dir<4; dir++ ) msgp[dir] += K;
    }


    double energy = 0.0;
    int i = 0;
    const float *labelp = label;
    for ( int y=0; y<height; y++ ) {
      for ( int x=0; x<width; x++ ) {
        energy += D[i*K+result[i]];
        // UP:
        int d = 0;
        if ( y > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist( lp0, lp1, dim, d ) * lambda;
        }


        // LEFT:
        d = 1;
        if ( x > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist( lp0, lp1, dim, d ) * lambda;
        }

        i++;
        labelp += K * dim;
      }
    }

    return energy;
  }
  */

  // Kernel Function for updating the result (agentNum = # pixels)
  /*
    __global__ void UpdateResult_agent_float( int agentNum,
    float *D,
    float *msg[4],
    int K,
    int *result  )
    {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
    float *Dp = D + idx * K;
    float *msgp[4];
    for ( int dir=0; dir<4; dir++ ) msgp[dir] = msg[dir] + idx * K;
    result[idx] = 0;
    floating min = 0;
    for ( int k=0; k<K; k++ ) {
    floating sum = Dp[k];
    for ( int dir=0; dir<4; dir++ ) {
    sum += msgp[dir][k];
    }

    if ( 0 == K ) {
    min = sum;
    } else if ( sum < min ) {
    min = sum;
    result[idx] = k;
    }
    }
    }
    }
  */

  
  double UpdateEnergy( const float *D, const float *label, int height, int width, int K,
                       int dim, float lambda, int *result )
  {

    // constants
    const int inc[4] = { -width, -1, width, 1 };
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};

    
    double energy = 0.0;
    int i = 0;
    const float *labelp = label;
    for ( int y=0; y<height; y++ ) {
      for ( int x=0; x<width; x++ ) {
        energy += D[i*K+result[i]];
        
        // UP:
        int d = 0;
        if ( y > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist( lp0, lp1, dim, d ) * lambda;
        }
        

        // LEFT:
        d = 1;
        if ( x > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist( lp0, lp1, dim, d ) * lambda;
        }

        i++;
        labelp += K * dim;
      }
    }

    return energy;
  }

  



  // Kernel Function for loopy belief propagation (agentNum = width/height)
  __global__ void LoopyBP_agent_float( int agentNum, // number of agents needes
                                       float *D,
                                       float *label,
                                       float* msg,
                                       int *begins,
                                       int *ends,
                                       int K,
                                       float lambda,
                                       int dim,
                                       int dir,
                                       int inc,
                                       int incK,
                                       int incDim,
                                       int area,
                                       float *buf_h )
  {
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      // agent initialization
      int opp = (dir+2) & 3;
      //      float *h = buf_h + idx * K;
      float *Dp = D + begins[idx] * K;
      float *labelp = label + begins[idx] * K * dim;

      for ( int i=begins[idx]; i!=ends[idx]; i+=inc, Dp+=incK, labelp+=incDim  ) {
        for ( int k=0; k<K; k++ ) {
          if ( i * K + k > 320 * K ) {
            printf( "%d\n", i*K+k );
          }
          buf_h[i*K+k] = Dp[k];
          for ( int j=0; j<4; j++ ) {
            if ( j != opp ) {
              buf_h[i*K+k] += msg[ j * area * K + i * K + incK ];
            }
          }
        }
        
        for ( int k=0; k<K; k++ ) {
          float min = 0.0;
          for ( int k0=0; k0<K; k0++ ) {
            const float *lp0 = labelp + k0 * dim;
            const float *lp1 = labelp + incDim + dim * k;
            float value = dist( lp0, lp1, dim, dir ) * lambda + buf_h[ i * K + k0 ];
            if ( 0 == k0 || value < min ) min = value;
          }
          msg[ dir * area * K + ( i + inc ) * K + k ] = min;
        }
      } // end for i
    }
  }
  
  
  double LoopyBP( const float *D, // The distance term for each pixel, h x w
                  const float *label,
                  // the label vectors for each candidates of each pixel, h x w x K x dim
                  const int height,
                  const int width,
                  const int K,
                  const int dim,
                  int *result,
                  Options options, float* msgBuf )
  {
    const int inc[4] = { -width, -1, width, 1 };
    const int incK[4] = {-width*K,-K,width*K,K};
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};
    const int order[4] = {2,0,3,1}; // DOWN, UP, RIGHT, LEFT
    const int area = height * width;

    
    // Make sure that the message storage is allocated
    float *buf = msgBuf;
    if ( NULL == buf ) {
      // Message buffer is not provided externally
      buf = new float[4*width*height*K];
    }

    



    // Initialization of device memory
    // Data term array
    float *devD = NULL;
    hipMalloc( (void**) &devD, sizeof(float) * area * K );
    hipMemcpy( devD, D, sizeof(float) * area * K, hipMemcpyHostToDevice );

    // Labels array
    float *devLabel = NULL;
    hipMalloc( (void**) &devLabel, sizeof(float) * area * K * dim );
    hipMemcpy( devLabel, label, sizeof(float) * area * K * dim, hipMemcpyHostToDevice );
    
    // Messages
    // After these lines, msg[0] = up, msg[1] = left, msg[2] = right, msg[3] = down
    float *devMsg = NULL;
    hipMalloc( (void**) &devMsg, sizeof(float) * area * K * 4 );
    hipMemset( devMsg, 0, sizeof(float) * area * K * 4 );
    

    // begins and ends
    int longer = height > width ? height : width;
    int *devBegins = NULL;
    int *devEnds = NULL;
    int begins[longer];
    int ends[longer];
    hipMalloc( (void**) &devBegins, sizeof(int) * longer );
    hipMalloc( (void**) &devEnds, sizeof(int) * longer );

    // h buffer
    float *devBufH = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devBufH, sizeof(float) * longer * K ) );
  
    // Result
    int *devResult = NULL;
    hipMalloc( (void**) &devResult, sizeof(int) * area * K );




    double energy(0);

    for ( int iter=0; iter<options.maxIter; iter++ ) {
      for ( int dirID=0; dirID<4; dirID++ ) {
        int dir = order[dirID];

        // fill in begins and ends
        int agentNum = ( 0 == ( dir & 1 ) ) ? width : height;
        int range = ( 0 == ( dir & 1 ) ) ? height * width : width;
        int stride = ( 0 == ( dir & 1 ) ) ? 1 : width;
        for ( int scan=0; scan<agentNum; scan++ ) {
          begins[scan] = scan * stride;
          ends[scan] = begins[scan] + range;
        }
        hipMemcpy( devBegins, begins, sizeof(int) * agentNum, hipMemcpyHostToDevice );
        hipMemcpy( devEnds, ends, sizeof(int) * agentNum, hipMemcpyHostToDevice );

        // Call Kernel Function 1
        LoopyBP_agent_float<<<(agentNum/32)+1,32>>>( agentNum, devD, devLabel,
                                                            devMsg, devBegins, devEnds,
                                                            K, options.lambda, dim, dir, inc[dir],
                                                            incK[dir], incDim[dir],
                                                            area,
                                                            devBufH );

        
        // debugging:
        float *tmpMsg = new float[area * K * 4];
        HANDLE_ERROR( hipMemcpy( tmpMsg, devMsg, sizeof(float) * area * K * 4, hipMemcpyDeviceToHost ) );
        float *loadMsg = new float[area * K * 4];
        FILE *in = fopen( "debug.dat", "r" );
        fread( loadMsg, sizeof(float), area * K * 4, in );
        fclose( in );
        for ( int l=0; l<area*K*4; l++ ) {
          if ( fabsf( tmpMsg[l] - loadMsg[l] ) > 1e-5 ) {
            printf( "%d: load(%.5f) vs gen(%.5f)\n", l, loadMsg[l], tmpMsg[l] );
            char ch;
            scanf( "%c", &ch );
          }
        }
        delete[] loadMsg;
        delete[] tmpMsg;
        printf( "debugging done\n" );
        exit( -1 );
        // end debugging


        
        UpdateResult_float_agent<<<(width*height)/32+1,32>>>( width * height,
                                                              devD,
                                                              devLabel,
                                                              devMsg,
                                                              devResult,
                                                              K,
                                                              dim,
                                                              options.lambda );


        hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost );
        
        energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
        
        NormalizeMessages_float_agent<<<(width*height)/32+1,32>>>( width * height,
                                                                   devMsg,
                                                                   K );

        
        
        /*
            __global__ void LoopyBP_agent_float( int agentNum, // number of agents needes
                                       float *D,
                                       float *label,
                                       float* msg[4],
                                       int *begins,
                                       int *ends,
                                       int K,
                                       float lambda,
                                       int dim,
                                       int dir,
                                       int inc,
                                       int incK,
                                       int incDim,
                                       int area,
                                       float *buf_h )

        */


        if ( 1 <= options.verbose ) {
          printf( "Iteration %d: energy = %.5lf\n", iter, energy );
        }

      } // end for dir
    }

    for ( int i=0; i<10; i++ ) {
      printf( "result[%d] = %d\n", i, result[i] );
    }
    return energy;
  }
};



