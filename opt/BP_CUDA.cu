#include "hip/hip_runtime.h"
/*********************************************************************************
 * File: BP_CUDA.cu
 * Description: Belief Propagation for Ising Model, CUDA parallelized version
 * by BreakDS, @ University of Wisconsin-Madison, Fri Aug 31 15:25:19 CDT 2012
 *********************************************************************************/

#include "BP_CUDA.h"
#include <hip/hip_runtime.h>
#include "../utils/cuda_aux.hpp"





int host_patch_side;

  

namespace optimize_cuda
{

  __constant__ int g_inc[4];
  __constant__ float g_coeff[6];
  __constant__ int g_patch_side;

  
  // This function calculate the distance between vector
  // ( a0, ..., a5 ) and ( b0, ..., b5 )
  // direction should be specified

  // __device__ inline float dist_device( const float a0, const float a1,
  //                                      const float a2, const float a3,
  //                                      const float a4, const float a5,
  //                                      const float b0, const float b1,
  //                                      const float b2, const float b3,
  //                                      const float b4, const float b5,
  //                                      int direction )
  // {
  //   float tmp;

  //   // [0] = image index
  //   tmp = fabsf( b0 - a0 );
  //   if ( tmp > 0.5 ) return 150000.0;

  //   // [1], [2] = dy, dx (rotation representation)
  //   tmp = fabsf( a1 - b1 ) + fabsf( a2 - b2  );
  //   if ( tmp > 1.0 ) {
  //     return 150000.0;
  //   }

      
  //   // [4],[5] = spatial distance
  //   // Should be compensated by the rotation (dy,dx)
  //   // Note that dy = b1 dx = b2 by definition

  //   float ay(a4), ax(a5);
    
  //   if ( 0 == direction ) {
  //     ay -= b2;
  //     ax -= b1;
  //   } else if ( 1 == direction ) {
  //     ay += b1;
  //     ax -= b2;
  //   } else if ( 2 == direction ) {
  //     ay += b2;
  //     ax += b1;
  //   } else if ( 3 == direction ) {
  //     ay -= b1;
  //     ax += b2;
  //   }
    
    
    
      
  //   tmp = fabsf( ay - b4 ) + fabsf( ax - b5 );
  //   if ( tmp > g_patch_side ) {
  //     return 150000.0;
  //   }
      
  //   // [1],[2] = spatial distance
    
  //   float sum = tmp * g_coeff[4];

  //   sum += fabsf( b1 - a1 ) * g_coeff[1];
  //   sum += fabsf( b2 - a2 ) * g_coeff[2];
  //   sum += fabsf( b3 - a3 ) * g_coeff[3];

    
  //   return sum;
  // }

  // // The host version of dist_device()
  // inline float dist_host( const float a0, const float a1,
  //                         const float a2, const float a3,
  //                         const float a4, const float a5,
  //                         const float b0, const float b1,
  //                         const float b2, const float b3,
  //                         const float b4, const float b5,
  //                         int direction )
  // {
  //   const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
  //   float tmp;

  //   // [0] = image index
  //   tmp = fabsf( b0 - a0 );
  //   if ( tmp > 1.0 ) return 150000.0;

  //   // [1], [2] = dy, dx (rotation representation)
  //   tmp = fabsf( a1 - b1 ) + fabsf( a2 - b2  );
  //   if ( tmp > 1.0 ) {
  //     return 150000.0;
  //   }

      
  //   // [4],[5] = spatial distance
  //   // Should be compensated by the rotation (dy,dx)
  //   // Note that dy = b1 dx = b2 by definition

  //   float ay(a4), ax(a5);

  //   if ( 0 == direction ) {
  //     ay -= b2;
  //     ax -= b1;
  //   } else if ( 1 == direction ) {
  //     ay += b1;
  //     ax -= b2;
  //   } else if ( 2 == direction ) {
  //     ay += b2;
  //     ax += b1;
  //   } else if ( 3 == direction ) {
  //     ay -= b1;
  //     ax += b2;
  //   }
    
    
    
      
  //   tmp = fabsf( ay - b4 ) + fabsf( ax - b5 );
  //   if ( tmp > host_patch_side ) {
  //     return 150000.0;
  //   }
      
  //   // [1],[2] = spatial distance
    
  //   float sum = tmp * coeff[4];

  //   sum += fabsf( b1 - a1 ) * coeff[1];
  //   sum += fabsf( b2 - a2 ) * coeff[2];
  //   sum += fabsf( b3 - a3 ) * coeff[3];

    
  //   return sum;
  // }



  __device__ inline float dist_device( const float a0, const float a1,
                                       const float a2, const float a3,
                                       const float a4, const float a5,
                                       const float b0, const float b1,
                                       const float b2, const float b3,
                                       const float b4, const float b5 )
  {
    float tmp;

    // [0] = image index
    tmp = fabsf( b0 - a0 );
    if ( tmp > 0.5 ) return 150000.0;

    tmp = fabsf( a1 - b1 ) + fabsf( a2 - b2  );
    if ( tmp > 1.5 ) {
      return 150000.0;
    }

    tmp = 0.0f;
    
    tmp += fabsf( b1 - a1 ) * g_coeff[1];
    tmp += fabsf( b2 - a2 ) * g_coeff[2];
    tmp += fabsf( b4 - a4 ) * g_coeff[4];
    tmp += fabsf( b5 - a5 ) * g_coeff[5];

    return tmp;
  }

  // The host version of dist_device()
  inline float dist_host( const float a0, const float a1,
                          const float a2, const float a3,
                          const float a4, const float a5,
                          const float b0, const float b1,
                          const float b2, const float b3,
                          const float b4, const float b5 )
  {
    const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
    float tmp;

    // [0] = image index
    tmp = fabsf( b0 - a0 );
    if ( tmp > 1.0 ) return 150000.0;

    tmp = fabsf( a1 - b1 ) + fabsf( a2 - b2  );
    if ( tmp > 1.5 ) {
      return 150000.0;
    }

    tmp = 0.0f;
    
    tmp += fabsf( b1 - a1 ) * coeff[1];
    tmp += fabsf( b2 - a2 ) * coeff[2];
    tmp += fabsf( b4 - a4 ) * coeff[4];
    tmp += fabsf( b5 - b5 ) * coeff[5];

    return tmp;
  }

  

  // normalize messages of a node (pixel) with the same direction (sum to 0)
  // agentNum: 4 * #pixel (4 = number of directions)
  // msg: the message array [dir(4), pixel(#pixel), k(K)]
  // K: # of candidate labels
  __global__ void NormalizeMsg_device( const int agentNum, 
                                       float *msg,
                                       int K )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      float offset = 0.0f;
      for ( int k=0; k<K; k++ ) {
        offset += msg[ idx * K + k ];
      }
      offset /= K;
      for ( int k=0; k<K; k++ ) {
        msg[ idx * K + k ] -= offset;
      }
    }
  }

  
  // Update the result array (one cadidate label id for each pixel)
  // agentNum: # of pixel
  // D: self distance matrix [pixel(# of pixels), k(K)];
  // msg: the message array [dir(4), pixel(# of pixels), k(K)]
  // result: the result array [pixel(# of pixels)]
  // K: # of labels per pixel
  __global__ void UpdateResult_device( const int agentNum, const float *D,
                                       const float* msg,
                                       int *result, int K )
  {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if ( idx < agentNum ) {
      result[idx] = 0;
      float min = 0.0f;
      for ( int k=0; k<K; k++ ) {
        float sum = D[idx*K+k];
        for ( int dir=0; dir<4; dir++ ) {
          sum += msg[ ( dir * agentNum + idx ) * K + k ];
        }

        if ( 0 == k ) {
          min = sum;
        } else if ( sum < min ) {
          min = sum;
          result[idx] = k;
        }
      }
    }
  }

  

  double UpdateEnergy( const float *D, const float *label, int height, int width, int K,
                       int dim, float lambda, int *result )
  {

    // constants
    const int inc[4] = { -width, -1, width, 1 };
    const int incDim[4] = {-width*K*dim,-K*dim,width*K*dim,K*dim};
    
    
    double energy = 0.0;
    int i = 0;
    const float *labelp = label;
    for ( int y=0; y<height; y++ ) {
      for ( int x=0; x<width; x++ ) {
        energy += D[i*K+result[i]];
        
        // UP:
        int d = 0;
        if ( y > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist_host( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                               lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5] ) * lambda;
        }
        

        // LEFT:
        d = 1;
        if ( x > 0 ) {
          const float *lp0 = labelp + result[i] * dim;
          const float *lp1 = labelp + incDim[d] + result[i+inc[d]] * dim;
          energy += dist_host( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                               lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5] ) * lambda;
        }

        i++;
        labelp += K * dim;
      }
    }

    return energy;
  }

  
  // Main Kernel Function, which updates messages
  // Currently this function takes 82.3% of the total cuda computation time
  // 
  // agentNum: columns or rows, depnding on the direction
  // D: self distance matrix [pixel(# of pixels), k(K)];
  // msg: the message array [dir(4), pixel(#pixel), k(K)]
  // distance: inter-pixel label distance matrix [k(K), dir(4), pixel(# of pixels)]
  // begins: array [pixel(agentNum)], where begins[i] is the first pixel of agent i
  // ends: array [pixel(agentNum)], where ends[i] is the last (exluded) pixel of agent i
  // K: # of candidate labels
  // lambda: coeffiecient of the inter-pixel label distance term
  // dim: dimension of a label
  // dir: current propagation direction
  // inc: pixel increment in this direction
  // incK: pixel increment, K times as larger as inc
  // area: # of pixels
  __global__ void UpdateMessage_device( int agentNum, // number of agents needes
                                        float *D,
                                        float* msg,
                                        float* distance,
                                        int *begins,
                                        int *ends,
                                        int K,
                                        float lambda,
                                        int dim,
                                        int dir,
                                        int inc,
                                        int incK,
                                        int area )
  {
    
    extern __shared__ float buf[];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if ( idx < agentNum ) {
      int k = threadIdx.y;
      float *h = buf;
      int opp = (dir+2) & 3;
      float *Dp = D + begins[idx] * K;

      int end = ends[idx];
      for ( int i=begins[idx]; i!=end; i+=inc, Dp+=incK ) {
        
        h[ k * blockDim.x + threadIdx.x ] = Dp[k];
        for ( int j=0; j<4; j++ ) {
          if ( j != opp ) {
            h[ k * blockDim.x + threadIdx.x ] += msg[ j * area * K + i * K + k ];
          }
        }
        __syncthreads();

        float min = distance[ ( k * 4 + dir ) * area  + i ] * lambda + h[ threadIdx.x ];


        for ( int k0=1; k0<K; k0++ ) {
          __syncthreads();
          float value = distance[ ( ( k0 * K + k ) * 4 + dir ) * area  + i ] * lambda +
            h[ k0 * blockDim.x + threadIdx.x];
          if ( value < min ) min = value;
        }
        msg[ dir * area * K + ( i + inc ) * K + k ] = min;
        
      } // end for i
    }
  }



  // Precompute the distance matrix that will be heavily used later
  // area: # of pixels, also serve as the agentNum
  // label: the label matrix [pixel(area), k(K), component(dim)]
  // distance: inter-pixel label distance matrix [k(K), dir(4), pixel(# of pixels)]
  // dim: dimension of each label
  // K: # of candidate labels per pixel
  __global__ void Precomputing_agent_float( int area,
                                            float *label,
                                            float *distance,
                                            int dim,
                                            int K )
  {
    int pixel = blockDim.x * blockIdx.x + threadIdx.x;
    if ( pixel < area ) {
      int dir = blockIdx.y;
      // int k0 = threadIdx.y;
      // int k1 = threadIdx.z;
      
      int idx = ( ( threadIdx.y * K + threadIdx.z ) * 4 + dir ) * area + pixel;
      // overflow checking
      if ( pixel + g_inc[dir] < 0 || pixel + g_inc[dir] >= area ) {
        return ;
      }
      
      float *lp0 = label + ( pixel * K + threadIdx.y ) * dim;
      float *lp1 = label + ( ( pixel + g_inc[dir] ) * K + threadIdx.z ) * dim;
      
      distance[idx] = dist_device( lp0[0], lp0[1], lp0[2], lp0[3], lp0[4], lp0[5],
                                   lp1[0], lp1[1], lp1[2], lp1[3], lp1[4], lp1[5] );
    }
  }




  
  
  double LoopyBP( const float *D, 
                  const float *label,
                  const int height,
                  const int width,
                  const int K,
                  const int dim,
                  int *result,
                  Options options, float* msgBuf )
  {

    
    
    const float coeff[6] = { 0.0, 30.0, 30.0, 10.0, 1.0, 1.0 };
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( optimize_cuda::g_coeff), &coeff, sizeof(float) * 6, 0, hipMemcpyHostToDevice ) );    

    int patchSide = 17;
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( optimize_cuda::g_patch_side), &patchSide, sizeof(int), 0, hipMemcpyHostToDevice ) );
    host_patch_side = patchSide;

    const int inc[4] = { -width, -1, width, 1 };
    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( optimize_cuda::g_inc), inc, sizeof(int) * 4, 0, hipMemcpyHostToDevice ) );

    const int incK[4] = {-width*K,-K,width*K,K};
    const int order[4] = {2,0,3,1}; // DOWN, UP, RIGHT, LEFT
    const int area = height * width;

    
    // Make sure that the message storage is allocated
    float *buf = msgBuf;
    if ( NULL == buf ) {
      // Message buffer is not provided externally
      buf = new float[4*width*height*K];
    }



    // Initialization of device memory
    // Data term array
    float *devD = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devD, sizeof(float) * area * K ) );
    HANDLE_ERROR( hipMemcpy( devD, D, sizeof(float) * area * K, hipMemcpyHostToDevice ) );

    // Labels array
    float *devLabel = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devLabel, sizeof(float) * area * K * dim ) );
    HANDLE_ERROR( hipMemcpy( devLabel, label, sizeof(float) * area * K * dim, hipMemcpyHostToDevice ) );
    
    // Messages
    // After these lines, msg[0] = up, msg[1] = left, msg[2] = right, msg[3] = down
    float *devMsg = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devMsg, sizeof(float) * area * K * 4 ) );
    HANDLE_ERROR( hipMemset( devMsg, 0, sizeof(float) * area * K * 4 ) );
    

    // begins and ends
    int longer = height > width ? height : width;
    int *devBegins = NULL;
    int *devEnds = NULL;
    int begins[longer];
    int ends[longer];
    HANDLE_ERROR( hipMalloc( (void**) &devBegins, sizeof(int) * longer ) );
    HANDLE_ERROR( hipMalloc( (void**) &devEnds, sizeof(int) * longer ) );

    // Result
    int *devResult = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devResult, sizeof(int) * area * K ) );

    // Intermediate Distance Calculation
    float *devDistance = NULL;
    HANDLE_ERROR( hipMalloc( (void**) &devDistance, sizeof(float) * K * K * 4 * area ) );

    dim3 precompute_grid( ( area + 1 ) / 8, 4 );
    dim3 precompute_block( 8, K, K );
    


    Precomputing_agent_float<<<precompute_grid, precompute_block>>>( area,
                                                                     devLabel,
                                                                     devDistance,
                                                                     dim,
                                                                     K );
    HANDLE_ERROR( hipDeviceSynchronize() );
    



    double energy(0);

    for ( int iter=0; iter<options.maxIter; iter++ ) {
      for ( int dirID=0; dirID<4; dirID++ ) {
        int dir = order[dirID];

        // fill in begins and ends
        int agentNum = ( 0 == ( dir & 1 ) ) ? width : height;
        if ( 0 == dir ) {
          // UP:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan + width * ( height - 1 );
            ends[scan] = scan;
          }
        } else if ( 1 == dir ) {
          // LEFT:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = width * scan + width - 1;
            ends[scan] = width * scan;
          }
        } else if ( 2 == dir ) {
          // DOWN:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan;
            ends[scan] = scan + width * ( height - 1 );
          }
        } else if ( 3 == dir ) {
          // RIGHT:
          for ( int scan=0; scan<agentNum; scan++ ) {
            begins[scan] = scan * width;
            ends[scan] = scan * width + width - 1;
          }
        }
        
        HANDLE_ERROR( hipMemcpy( devBegins, begins, sizeof(int) * agentNum, hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( devEnds, ends, sizeof(int) * agentNum, hipMemcpyHostToDevice ) );
        


        // call Kernel Function 1
        int blockSize = 8;
        dim3 blockDim( blockSize, K );
        int shMemSizePerBlock= blockSize * K * sizeof(float);
        UpdateMessage_device<<<(agentNum/blockSize)+1,blockDim, shMemSizePerBlock>>>( agentNum, 
                                                                                      devD,
                                                                                      devMsg,
                                                                                      devDistance,
                                                                                      devBegins, 
                                                                                      devEnds,
                                                                                      K, 
                                                                                      options.lambda, 
                                                                                      dim, dir, inc[dir],
                                                                                      incK[dir],
                                                                                      area );
        
        HANDLE_ERROR( hipDeviceSynchronize() );


        
        NormalizeMsg_device<<<(width*height*4)/128+1,128>>>( width * height * 4,
                                                             devMsg,
                                                             K );
        if ( 1 <= options.verbose ) {
          UpdateResult_device<<<(width*height+1)/64,64>>>( width * height,
                                                           devD,
                                                           devMsg,
                                                           devResult,
                                                           K );

          
          HANDLE_ERROR( hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost ) );
          energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
          printf( "Iteration %d: energy = %.5lf\n", iter, energy );
        }
        
        
        
      } // end for dir
      
    }

    UpdateResult_device<<<(width*height+1)/64,64>>>( width * height,
                                                     devD,
                                                     devMsg,
                                                     devResult,
                                                     K );

    HANDLE_ERROR( hipMemcpy( result, devResult, sizeof(int) * width * height, hipMemcpyDeviceToHost ) );
    energy = UpdateEnergy( D, label, height, width, K, dim, options.lambda, result );
    printf( "Final energy = %.5lf\n", energy );

    // Free Cuda Memory
    if ( devD ) HANDLE_ERROR( hipFree( devD ) );
    if ( devLabel ) HANDLE_ERROR( hipFree( devLabel ) );
    if ( devMsg ) HANDLE_ERROR( hipFree( devMsg ) );
    if ( devBegins ) HANDLE_ERROR( hipFree( devBegins ) );
    if ( devEnds ) HANDLE_ERROR( hipFree( devEnds ) );
    if ( devResult ) HANDLE_ERROR( hipFree( devResult ) );
    if ( devDistance ) HANDLE_ERROR( hipFree( devDistance ) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    
    return energy;
  }
};



